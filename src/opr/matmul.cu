#include "hip/hip_runtime.h"
#include "opr_utils.h"



__global__ void kernel_matmul(float *out, TensorFormat *out_format,
        float *a, TensorFormat *a_format,
        float *b, TensorFormat *b_format) {
    const size_t BLOCK_DIM = 32;
    size_t inner_x = threadIdx.x;
    size_t inner_y = threadIdx.y;
    size_t batch_idx = blockIdx.z;
    size_t outer_x = blockIdx.x;
    size_t outer_y = blockIdx.y;
    size_t idx_x = outer_x * BLOCK_DIM + inner_x;
    size_t idx_y = outer_y * BLOCK_DIM + inner_y;

    size_t batch_stride_a = a_format->shape[0] == 1 ? 0: a_format->strides[0];
    size_t batch_stride_b = b_format->shape[0] == 1 ? 0: b_format->strides[0];
    size_t batch_stride_o = out_format->shape[0] == 1 ? 0: out_format->strides[0];

    size_t reduction_dim = a_format->shape[2];

    __shared__ float buf_a[32][32], buf_b[32][32];

    float ans = 0.f;
    for(size_t r=0; r < reduction_dim; r+= BLOCK_DIM) {
        size_t idx_a = batch_stride_a * batch_idx + a_format->strides[1] * idx_x + a_format->strides[2] * (r + inner_y);
        size_t idx_b = batch_stride_b * batch_idx + b_format->strides[2] * idx_y + b_format->strides[1] * (r + inner_x);
        
        if(idx_x < a_format->shape[1] && a_format->shape[2])
            buf_a[inner_x][inner_y] = a[idx_a/sizeof(float)];
        else
            buf_a[inner_x][inner_y] = 0.f;
        
        if(idx_x < b_format->shape[1] && b_format->shape[2])
            buf_b[inner_x][inner_y] = b[idx_b/sizeof(float)];
        else
            buf_b[inner_x][inner_y] = 0.f;
        __syncthreads();

        for(int i=0;i<BLOCK_DIM;i++) {
            ans += buf_a[inner_x][i] * buf_b[i][inner_y];
        }
        __syncthreads();
    }
    size_t out_idx = (batch_stride_o * batch_idx + out_format->strides[1] * idx_x + out_format->strides[2] * idx_y) / sizeof(float);
    if(idx_x<out_format->shape[1] && idx_y < out_format->shape[2])
        out[out_idx] = ans;
}

void print_shape(vector<size_t> &data) {
    for(size_t i: data)
        printf("%zu ", i);
    printf("\n");
}


shared_ptr<TensorStorage> matmul_op(const shared_ptr<TensorStorage> x, bool x_transpose, const shared_ptr<TensorStorage> &y, bool y_transpose) {
    vector<size_t> x_shape = x->m_shape;
    vector<size_t> y_shape = y->m_shape;
    vector<size_t> x_strides = x->m_strides;
    vector<size_t> y_strides = y->m_strides;

    // printf("matmul_op data 1 shape: ");
    // print_shape(x_shape);
    // printf("matmul_op data 1 strides: ");
    // print_shape(x_strides);
    // printf("matmul_op data 2 shape: ");
    // print_shape(y_shape);
    // printf("matmul_op data 2 strides: ");
    // print_shape(y_strides);

    bool x_extended = false;
    if(x_shape.size() == 2) {
        x_shape.insert(x_shape.begin(), 1);
        x_strides.insert(x_strides.begin(), x_strides[0]);
        x_extended = true;
    }
    bool y_extended = false;
    if(y_shape.size() == 2) {
        y_shape.insert(y_shape.begin(), 1);
        y_strides.insert(y_strides.begin(), y_strides[0]);
        y_extended = true;
    }
    if(x_transpose) {
        swap(x_shape[1], x_shape[2]);
        swap(x_strides[1], x_strides[2]);
    }
    if(y_transpose) {
        swap(y_shape[1], y_shape[2]);
        swap(y_strides[1], y_strides[2]);
    }
    assert(x_shape[2] == y_shape[1]);
    
    vector<size_t> output_shape;
    vector<size_t> output_strides;
    size_t output_size = 1;
    output_shape.push_back(max(x_shape[0], y_shape[0]));
    output_shape.push_back(x_shape[1]);
    output_shape.push_back(y_shape[2]);

    for(size_t i=0;i<output_shape.size();i++) {
        output_strides.push_back(output_size * sizeof(float));
        output_size *= output_shape[i];
    }
    float *res;
    hipMalloc(&res, sizeof(float) * output_size);

    TensorFormat *x_format = TensorFormat::make_cuda_tensor_format(x_shape, x_strides);
    TensorFormat *y_format = TensorFormat::make_cuda_tensor_format(y_shape, y_strides);
    TensorFormat *out_format = TensorFormat::make_cuda_tensor_format(output_shape, output_strides);

    // 
    int block_size = 32;
    dim3 threads(block_size, block_size);
    dim3 blocks((output_shape[1] + block_size - 1)/block_size, 
            (output_shape[2] +block_size - 1)/block_size, 
            output_shape[0]);
    
    kernel_matmul<<<blocks, threads>>>(res, out_format, x->m_data, x_format, y->m_data, y_format);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error\n");
    }
    x_format->release();
    y_format->release();
    out_format->release();
    if(x_extended && y_extended) {
        output_shape.erase(output_shape.begin());
        output_strides.erase(output_strides.begin());
    }
    // printf("output_size: %zu\n", output_size);
    // printf("matmul_op output shape: ");
    // print_shape(output_shape);
    // printf("matmul_op output strides: ");
    // print_shape(output_strides);
    return make_shared<TensorStorage>(res, output_size, output_shape, output_strides);
}


shared_ptr<TensorStorage> matmul_op(const shared_ptr<TensorStorage> x, const shared_ptr<TensorStorage> &y) {
    return matmul_op(x, false, y, false);
}


struct MatmulOpBackwarFunc: BackwardFunc {
    static std::shared_ptr<BackwardFunc> make(shared_ptr<GraphNode> x, shared_ptr<GraphNode> y){
        shared_ptr<BackwardFunc> func = make_shared<MatmulOpBackwarFunc>();
        func->m_input_nodes.push_back(x);
        func->m_input_nodes.push_back(y);
        return func;
    }

    void backward_func(shared_ptr<GraphNode> out_node) override {
        shared_ptr<TensorStorage> out_grad = out_node->m_grad_storage;
        shared_ptr<TensorStorage> x1 = m_saved_tensors[0];
        shared_ptr<TensorStorage> x2 = m_saved_tensors[1];
        
        shared_ptr<TensorStorage> g1 = matmul_op(out_grad, false, x2, true);
        shared_ptr<TensorStorage> g2 = matmul_op(x1, true, out_grad, false);
        
        m_input_nodes[0]->acc_grad(g1);
        m_input_nodes[1]->acc_grad(g2);
    }
};


Tensor matmul_op(Tensor &x, Tensor &y) {
    Tensor res = Tensor(matmul_op(x.m_storage, y.m_storage));
    if(x.m_need_grad || y.m_need_grad || x.m_require_grad || y.m_require_grad) {
        shared_ptr<GraphNode> x_node = x.graph_node();
        shared_ptr<GraphNode> y_node = y.graph_node();
        shared_ptr<GraphNode> out_node = res.graph_node();
        shared_ptr<BackwardFunc> func = MatmulOpBackwarFunc::make(x_node, y_node);
    
        func->m_saved_tensors.push_back(x.m_storage);
        func->m_saved_tensors.push_back(y.m_storage);
        
        out_node->set_backward_func(func);
        out_node->m_need_grad = true;
        res.m_need_grad = true;
    }
    return res;
}

namespace opr{

Tensor matmul(Tensor &x, Tensor &y) {
    return matmul_op(x, y);
}

}
